
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <vector>
#include <iostream>
#include <chrono>

class Estimation_CUDA
{
public:
	void DoMeanShift(bool* bImage, const int& rows, const int& cols,
		float* centers, const int& centerNums,
		const float& radius, const float& deadDist);

	void MemoryPreAllocate(const int& rows, const int& cols, const int& centerNums, const int& iterLimit);
private:
	bool* mK_bImage;
	int* mK_rows_Const;
	int* mK_cols_Const;

	float* mK_centers;
	int* mK_centerNums_Const;

	// x_start , x_end , y_start , y_end
	// [i*4 + 0], max i is centerNums
	int* mK_temp_range;
	// max i is centerNums
	float* mK_temp_n;
	// dot_x_sum , dot_y_sum
	// [i*2 + 0], max i is centerNums
	int* mK_temp_dots;
	float* mK_temp_dist;

	float* mK_radius_Const;
	float* mK_deadDist_Const;

	int* mK_IterLimit;

	void MemoryFree();
public:
	Estimation_CUDA(const int& GPUNumber = 0);
	~Estimation_CUDA();
};

__global__ void meanShiftKernal(bool* bImage, int* rows, int* cols,
	float* centers, int* centerNums,
	int* temp_range, int* dots, float* n, float* dist,
	float* radius, float* deadDist, int* iterLimit);


Estimation_CUDA::Estimation_CUDA(const int& GPUNumber)
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
}

Estimation_CUDA::~Estimation_CUDA()
{
	MemoryFree();

	hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}
}

void Estimation_CUDA::MemoryPreAllocate(const int& rows, const int& cols, const int& centerNums, const int& iterLimit)
{
	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

	//�޸� �Ҵ�
	hipMalloc(&mK_bImage, rows * cols * sizeof(bool));
	hipMalloc(&mK_rows_Const, sizeof(int));
	hipMalloc(&mK_cols_Const, sizeof(int));

	hipMalloc(&mK_centers, centerNums * 5 * sizeof(float));
	hipMalloc(&mK_centerNums_Const, sizeof(float));

	hipMalloc(&mK_temp_range, centerNums * 4 * sizeof(int));
	hipMalloc(&mK_temp_n, centerNums * sizeof(float));
	hipMalloc(&mK_temp_dots, centerNums * 2 * sizeof(int));
	hipMalloc(&mK_temp_dist, centerNums * sizeof(float));

	hipMalloc(&mK_IterLimit, sizeof(int));
	hipMemcpy(mK_IterLimit, &iterLimit, sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&mK_radius_Const, sizeof(float));
	hipMalloc(&mK_deadDist_Const, sizeof(float));

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	std::cout << "Time difference(mem allocate) = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;

}

void Estimation_CUDA::MemoryFree()
{
	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
	//GPU �޸� ����
	hipFree(mK_bImage);
	hipFree(mK_rows_Const);
	hipFree(mK_cols_Const);

	hipFree(mK_centers);
	hipFree(mK_centerNums_Const);

	hipFree(mK_temp_range);
	hipFree(mK_temp_dots);
	hipFree(mK_temp_n);
	hipFree(mK_temp_dist);

	hipFree(mK_IterLimit);

	hipFree(mK_radius_Const);
	hipFree(mK_deadDist_Const);

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	std::cout << "Time difference(mem free) = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;
}

void Estimation_CUDA::DoMeanShift(bool* bImage, const int& rows, const int& cols,
	float* centers, const int& centerNums,
	const float& radius, const float& deadDist)
{

	//�޸� �ʱ�ȭ
	hipMemcpy(mK_bImage, bImage, rows * cols * sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(mK_rows_Const, &rows, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(mK_cols_Const, &cols, sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(mK_centers, centers, centerNums * 5 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(mK_centerNums_Const, &centerNums, sizeof(float), hipMemcpyHostToDevice);

	//cudaMemcpy(K_temp_range, temp_range,centerNums * 4 * sizeof(int), cudaMemcpyHostToDevice);
	//cudaMemcpy(K_temp_n, temp_n, centerNums * sizeof(float), cudaMemcpyHostToDevice);
	//cudaMemcpy(K_temp_dots,temp_dots, centerNums * 2 * sizeof(int), cudaMemcpyHostToDevice);

	hipMemcpy(mK_radius_Const, &radius, sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(mK_deadDist_Const, &deadDist, sizeof(float),hipMemcpyHostToDevice);


	//���� �׸��� �� ������ ���� ����
	const int ThreadNum = 64;
	const int BlockNum = (centerNums / 64) +1;

	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
	//GPU Ŀ�� ����
	meanShiftKernal <<<BlockNum, ThreadNum>>>(mK_bImage, mK_rows_Const, mK_cols_Const,
		mK_centers, mK_centerNums_Const, 
		mK_temp_range, mK_temp_dots, mK_temp_n, mK_temp_dist,
		mK_radius_Const, mK_deadDist_Const, mK_IterLimit);


	

	//��� �Ϸ�� �޸� ȣ��Ʈ�� ����
	hipMemcpy(centers, mK_centers, centerNums * 5 * sizeof(float), hipMemcpyDeviceToHost);

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	std::cout << "Time difference(gpu kernel) = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;

}

__global__ void meanShiftKernal(bool* bImage, int* rows, int* cols,
	float* centers, int* centerNums, 
	int* temp_range, int* dots, float* n, float* dist,
	float* radius, float* deadDist, int* iterLimit)
{

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int iter = 0;
	temp_range[i * 4 + 0] = 0;
	temp_range[i * 4 + 1] = 0;
	temp_range[i * 4 + 2] = 0;
	temp_range[i * 4 + 3] = 0;

	dist[i] = 0.0f;

	dots[i * 2 + 0] = 0;
	dots[i * 2 + 1] = 0;

	n[i] = 0.0f;



	if (i < *centerNums)
	{
		while (true)
		{
			//initialize temp_ranges;
			temp_range[i * 4 + 0] = (int)(centers[i * 5 + 2] - *radius);
			temp_range[i * 4 + 1] = (int)(centers[i * 5 + 2] + *radius);
			temp_range[i * 4 + 2] = (int)(centers[i * 5 + 3] - *radius);
			temp_range[i * 4 + 3] = (int)(centers[i * 5 + 3] + *radius);


			if (temp_range[i * 4 + 0] < 0) temp_range[i * 4 + 0] = 0;
			if (temp_range[i * 4 + 1] > *cols) temp_range[i * 4 + 1] = *cols;
			if (temp_range[i * 4 + 2] < 0) temp_range[i * 4 + 2] = 0;
			if (temp_range[i * 4 + 3] > *rows) temp_range[i * 4 + 3] = *rows;
		
			dots[i * 2 + 0] = 0;
			dots[i * 2 + 1] = 0;
			n[i] = 0.0f;
			for (int r = temp_range[i * 4 + 2]; r < temp_range[i * 4 + 3]; r++)
			{
				for (int c = temp_range[i * 4 + 0]; c < temp_range[i * 4 + 1]; c++)
				{

					if (bImage[r * (*cols) + c] == true)
					{
						dist[i] = hypotf(centers[i * 5 + 2] - c, centers[i * 5 + 3] - r);
						if (dist[i] < *radius)
						{
							dots[i * 2 + 0] += c;
							dots[i * 2 + 1] += r;
							n[i]++;
						}
					}
					
				}
			}
			
			if (n[i] > 0)
			{
				centers[i * 5 + 0] = dots[i * 2 + 0] / n[i];
				centers[i * 5 + 1] = dots[i * 2 + 1] / n[i];
			}
			

			dist[i] = hypotf(centers[i * 5 + 0] - centers[i * 5 + 2], centers[i * 5 + 1] - centers[i * 5 + 3]);
			if (dist[i] < *deadDist)
			{
				centers[i * 5 + 4] = n[i] / ((*radius) * (*radius) * (3.14159f));
				break;
			}
			else
			{
				centers[i * 5 + 2] = centers[i * 5 + 0];
				centers[i * 5 + 3] = centers[i * 5 + 1];
			}
			
			iter++;
			if (iter > * iterLimit)
			{
				centers[i * 5 + 4] = n[i] / ((*radius) * (*radius) * (3.14159f));
				break;
			}
			

		}
	}
	
	
}
